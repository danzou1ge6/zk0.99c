#include "./ntt_c_api.h"
#include "../../../ntt/src/runtime.cuh"
#include <hip/hip_runtime.h>

runtime::ntt_runtime<runtime::fifo> temp_runtime(1);

bool cuda_ntt(unsigned int *data, const unsigned int *omega, unsigned int log_n, FIELD field, bool inverse, bool process, const unsigned int * inv_n, const unsigned int * zeta, unsigned int start_n) {
    auto id = runtime::ntt_id{log_n, field, process, inverse};

    bool success = true;
    hipError_t first_err = hipSuccess;
    if (field == FIELD::PASTA_CURVES_FIELDS_FP) {
        CUDA_CHECK(hipHostRegister((void *)data, 8ull * sizeof(uint) * (1 << log_n), hipHostRegisterDefault));
    } else if (field == FIELD::HALO2CURVES_BN256_FR) {
        CUDA_CHECK(hipHostRegister((void *)data, 8ull * sizeof(uint) * (1 << log_n), hipHostRegisterDefault));
    } else {
        return false;
    }
    
    if (first_err == hipSuccess) try {
        auto ntt_kernel = temp_runtime.get_ntt_kernel(id, omega, inv_n, zeta);
        hipStream_t stream;
        CUDA_CHECK(hipStreamCreate(&stream));
        CUDA_CHECK(ntt_kernel->ntt(data, stream, start_n));
        CUDA_CHECK(hipStreamDestroy(stream));
    } catch(const char *msg) {
        std::cerr << msg << std::endl;
        success = false;
    }
    CUDA_CHECK(hipHostUnregister((void *)data));

    return success;
}

// bool cuda_coeff_to_extended(unsigned int *data, const unsigned int *omega, unsigned int log_n, FIELD field, const unsigned int * zeta, unsigned int **dev_ptr, unsigned int start_n, void ** stream) {
//     auto id = runtime::ntt_id{log_n, field, true, false};

//     bool success = true;
//     hipError_t first_err = hipSuccess;
//     hipStream_t *stm;
//     stm = (hipStream_t *)malloc(sizeof(hipStream_t));
//     *stream = stm;
//     CUDA_CHECK(hipStreamCreate((hipStream_t *) stm));
//     if (field == FIELD::PASTA_CURVES_FIELDS_FP) {
//         CUDA_CHECK(hipHostRegister((void *)data, 8ull * sizeof(uint) * (start_n), hipHostRegisterDefault));
//     } else if (field == FIELD::HALO2CURVES_BN256_FR) {
//         CUDA_CHECK(hipHostRegister((void *)data, 8ull * sizeof(uint) * (start_n), hipHostRegisterDefault));
//     } else {
//         return false;
//     }
    
//     if (first_err == hipSuccess) try {
//         auto ntt_kernel = temp_runtime.get_ntt_kernel(id, omega, nullptr, zeta);
//         CUDA_CHECK(ntt_kernel->ntt(data, *stm, start_n, dev_ptr));
//     } catch(const char *msg) {
//         std::cerr << msg << std::endl;
//         success = false;
//     }

//     return success;
// }