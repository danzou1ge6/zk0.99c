#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>
#include "cuda_api.h"

#define CUDA_CHECK(call)                                                                                             \
{                                                                                                                    \
    hipError_t err = call;                                                                                          \
    if (err != hipSuccess) {                                                                                        \
        std::cerr << "CUDA Error [" << __FILE__ << ":" << __LINE__ << "]: " << hipGetErrorString(err) << std::endl; \
        success = false;                                                                                             \
    }                                                                                                                \
}

bool cuda_device_to_host_sync(void *dst, const void *src, unsigned long size, const void* stream_ptr) {
    hipStream_t stream =  stream_ptr == nullptr ? 0 : *(hipStream_t*)stream_ptr;
    bool success = true;
    
    CUDA_CHECK(hipHostRegister(dst, size, hipHostRegisterDefault));
    CUDA_CHECK(hipMemcpyAsync(dst, src, size, hipMemcpyDeviceToHost, stream));
    CUDA_CHECK(hipStreamSynchronize(stream));
    CUDA_CHECK(hipHostUnregister(dst));

    return success;
}

bool cuda_unregister(void *src) {
    bool success = true;
    CUDA_CHECK(hipHostUnregister(src));
    return success;
}

void cpp_free(void * src) {
    free(src);
}

bool cuda_free(void * dev_ptr, const void *stream_ptr) {
    hipStream_t stream =  stream_ptr == nullptr ? 0 : *(hipStream_t*)stream_ptr;
    bool success = true;
    CUDA_CHECK(hipFreeAsync(dev_ptr, stream));
    return success;
}