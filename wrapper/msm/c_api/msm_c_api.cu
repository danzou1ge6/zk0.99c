#include "./msm_c_api.h"
#include "../../../msm/src/msm.cuh"
#include "../../../msm/src/bn254.cuh"
#include "../../../mont/src/bn254_scalar.cuh"

#include <hip/hip_runtime.h>
#include <hipcub/hipcub.hpp>

using mont::u32;
using bn254::Point;
using bn254::PointAffine;
using bn254_scalar::Number;
using bn254_scalar::Element;


bool cuda_msm(unsigned int len, const unsigned int* scalers, const unsigned int* points, unsigned int* res) {

    bool success = true;
    
    hipHostRegister((void*)scalers, len * sizeof(Number), hipHostRegisterDefault);
    hipHostRegister((void*)points, len * sizeof(PointAffine), hipHostRegisterDefault);

    using Config = msm::MsmConfig<255, 16, 16, false>;
    u32 batch_size = 1;
    u32 batch_per_run = 1;
    u32 parts = 2;
    u32 stage_scalers = 2;
    u32 stage_points = 2;

    std::array<u32*, Config::n_precompute> h_points;
    h_points[0] = (u32*)points;
    for (u32 i = 1; i < Config::n_precompute; i++) {
        hipHostAlloc(&h_points[i], len * sizeof(PointAffine), hipHostMallocDefault);
    }

    
    std::vector<u32*> scalers_batches;
    for (int i = 0; i < batch_size; i++) {
        scalers_batches.push_back((u32*)scalers);
    }

    std::vector<Point> r(batch_size);

    std::vector<u32> cards;
    int card_count;
    hipGetDeviceCount(&card_count);
    for (int i = 0; i < card_count; i++) {
        cards.push_back(i);
    }

    msm::MultiGPUMSM<Config, Number, Point, PointAffine> msm_solver(len, batch_per_run, parts, stage_scalers, stage_points, cards);

    // std::cout << "start precompute" << std::endl;

    hipStream_t stream;
    hipStreamCreate(&stream);
    msm::MSMPrecompute<Config, Point, PointAffine>::precompute(len, h_points);
    msm_solver.set_points(h_points);

    // std::cout << "Precompute done" << std::endl;
    msm_solver.alloc_gpu();
    // std::cout << "Alloc GPU done" << std::endl;
    hipEvent_t start, stop;
    float elapsedTime = 0.0;

    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    msm_solver.msm(scalers_batches, r);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);
    // std::cout << "Run done" << std::endl;

    hipStreamDestroy(stream);

    // for (int i = 0; i < batch_size; i++) {
    //     std::cout << r[i].to_affine() << std::endl;
    // }

    // std::cout << "Total cost time:" << elapsedTime << std::endl;
    hipEventDestroy(start);
    hipEventDestroy(stop);

    hipHostUnregister((void*)scalers);
    hipHostUnregister((void*)points);
    for (u32 i = 1; i < Config::n_precompute; i++) {
        hipHostFree(h_points[i]);
    }

    auto r_affine = r[0].to_affine();

    auto x = r_affine.x;
    auto y = r_affine.y;
    auto z = Element::one();

    if (r_affine.is_identity()) { // identity
        x = Element::zero();
        y = Element::one();
        z = Element::zero();
    }

    for(int i=0;i<Element::LIMBS;++i) {
        res[i] = x.n.limbs[i];
    }
    for(int i = 0; i < Element::LIMBS; ++i) {
        res[i+Element::LIMBS] = y.n.limbs[i];
    }
    for(int i = 0; i < Element::LIMBS; ++i) {
        res[i + Element::LIMBS * 2] = z.n.limbs[i];
    }

    return success;
}