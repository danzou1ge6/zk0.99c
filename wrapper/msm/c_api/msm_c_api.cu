#include "./msm_c_api.h"
#include "../../../msm/src/msm_radix_sort.cuh"
#include "../../../msm/src/bn254.cuh"
#include "../../../mont/src/bn254_scalar.cuh"

#include <hip/hip_runtime.h>
#include <hipcub/hipcub.hpp>

using mont::u32;
using bn254::Point;
using bn254::PointAffine;
using bn254_scalar::Element;


bool cuda_msm(unsigned int len, const unsigned int* scalers, const unsigned int* points, unsigned int* res) {

    bool success = true;
    
    hipHostRegister((void*)scalers, len * sizeof(Element), hipHostRegisterDefault);
    hipHostRegister((void*)points, len * sizeof(PointAffine), hipHostRegisterDefault);

    u32 *d_points, *h_points_precompute, head;

    hipStream_t stream;
    hipStreamCreate(&stream);

    msm::precompute<msm::MsmConfig<>>((u32*)points, len, d_points, h_points_precompute, head, stream);

    hipEvent_t start, stop;
    float elapsedTime = 0.0;

    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    Point r;
    msm::run<msm::MsmConfig<>>((u32*)scalers, d_points, len, r, h_points_precompute, head, stream);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);

    hipFree(d_points);

    hipStreamDestroy(stream);

    std::cout << "Total cost time:" << elapsedTime << std::endl;
    hipEventDestroy(start);
    hipEventDestroy(stop);

    hipHostUnregister((void*)scalers);
    hipHostUnregister((void*)points);
    hipHostFree(h_points_precompute);
    hipFree(d_points);

    auto r_affine = r.to_affine();

    for(int i=0;i<Element::LIMBS;++i) {
      res[i] = r_affine.x.n.limbs[i];
    }
    for(int i = 0; i < Element::LIMBS; ++i) {
      res[i+Element::LIMBS] = r_affine.y.n.limbs[i];
    }
    for(int i = 0; i < Element::LIMBS; ++i) {
      res[i + Element::LIMBS * 2] = Element::one().n.limbs[i];
    }

    return success;
}