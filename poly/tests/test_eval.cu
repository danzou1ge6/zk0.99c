#include "hip/hip_runtime.h"
#define DOCTEST_CONFIG_IMPLEMENT_WITH_MAIN
#include <doctest/doctest.h>
#include "../src/poly_eval.cuh"
#include "../../mont/src/bn254_fr.cuh"

typedef bn254_fr::Element Field;

Field* gen_poly(uint len) {
    Field* poly = new Field[len];
    for (uint i = 0; i < len; i++) {
        poly[i] = Field::host_random();
    }
    return poly;
}

Field eval_cpu(Field * poly, Field x, uint len) {
    Field res = Field::zero();
    for (uint i = 0; i < len; i++) {
        res = res + poly[i] * x.pow(i);
    }
    return res;
}

TEST_CASE("Naive eval") {
    auto len = 1 << 24;
    auto poly = gen_poly(len);
    auto x = Field::host_random();
    uint* poly_d;
    hipMalloc(&poly_d, len * Field::LIMBS * sizeof(uint));
    hipMemcpy(poly_d, poly, len * Field::LIMBS * sizeof(uint), hipMemcpyHostToDevice);
    uint *res_d;
    hipMalloc(&res_d, Field::LIMBS * sizeof(uint));
    hipEvent_t start, end;
    hipEventCreate(&start);
    hipEventCreate(&end);
    hipEventRecord(start);
    poly::NaiveEval(poly_d, poly_d, res_d, x, len, 0);
    hipEventRecord(end);
    hipEventSynchronize(end);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, end);
    std::cout << "Time: " << milliseconds << "ms" << std::endl;
    Field res;
    hipMemcpy(&res, res_d, Field::LIMBS * sizeof(uint), hipMemcpyDeviceToHost);
    auto res_cpu = eval_cpu(poly, x, len);
    CHECK(res == res_cpu);
    hipFree(poly_d);
    hipFree(res_d);
    delete [] poly;
}

TEST_CASE("eval") {
    auto len = 1 << 24;
    auto poly = gen_poly(len);
    auto x = Field::host_random();
    uint* poly_d;
    hipMalloc(&poly_d, len * Field::LIMBS * sizeof(uint));
    hipMemcpy(poly_d, poly, len * Field::LIMBS * sizeof(uint), hipMemcpyHostToDevice);
    uint *res_d;
    hipMalloc(&res_d, Field::LIMBS * sizeof(uint));
    uint *temp_buf;
    hipMalloc(&temp_buf, len * Field::LIMBS * sizeof(uint));
    hipEvent_t start, end;
    hipEventCreate(&start);
    hipEventCreate(&end);
    hipEventRecord(start);
    poly::Eval(poly_d, temp_buf, res_d, x, len, 0);
    hipEventRecord(end);
    hipEventSynchronize(end);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, end);
    std::cout << "Time: " << milliseconds << "ms" << std::endl;
    Field res;
    hipMemcpy(&res, res_d, Field::LIMBS * sizeof(uint), hipMemcpyDeviceToHost);
    auto res_cpu = eval_cpu(poly, x, len);
    CHECK(res == res_cpu);

    hipFree(poly_d);
    hipFree(res_d);
    delete [] poly;
}