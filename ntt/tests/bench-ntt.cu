#include "../src/self_sort_in_place_ntt.cuh"
#include "../../mont/src/bn254_fr.cuh"
#include "../../mont/src/mnt4753_fr.cuh"

using namespace ntt;
// typedef mnt4753_fr::Element Field;
typedef bn254_fr::Element Field;

int main () {
    for (int k = 20; k <= 28; k += 2) {
        auto omega = Field::host_random();
        auto config = self_sort_in_place_ntt<Field>::SSIP_config();
        
        config.max_threads_stage1_log = 8;
        config.max_threads_stage2_log = 8;
        self_sort_in_place_ntt<Field> ntt(reinterpret_cast<u32*>(&omega), k, false, 1, false, false, nullptr, nullptr, config);
        ntt.to_gpu();
        
        Field *data, *data_d;
        hipMalloc(&data_d, (1ll << k) * sizeof(Field));
        data = (Field*)malloc((1ll << k) * sizeof(Field));
        for (int i = 0; i < (1ll << k); i++) {
            data[i] = Field::host_random();
        }
        hipMemcpy(data_d, data, (1ll << k) * sizeof(Field), hipMemcpyHostToDevice);

        // warm up, because the jit compilation is slow
        for (int i = 0; i < 10; i++) ntt.ntt(reinterpret_cast<u32*>(data_d), 0, 0, true);

        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);

        for (int i = 0; i < 10; i++) {
            hipEventRecord(start);
            ntt.ntt(reinterpret_cast<u32*>(data_d), 0, 0, true);
            hipEventRecord(stop);
            hipEventSynchronize(stop);
            float milliseconds = 0;
            hipEventElapsedTime(&milliseconds, start, stop);
            printf("k = %d, time = %f ms\n", k, milliseconds);
        }
        
        free(data);
        hipFree(data_d);
    }
    return 0;
}
