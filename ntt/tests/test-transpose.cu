#include "hip/hip_runtime.h"
#include "../src/inplace_transpose/cuda/transpose.cuh"
#include <cstring>
#include <hip/hip_runtime.h>
#define IDX(i, j, n) ((i) * (n) + (j))
struct chunk {
    int data[8];
    __device__ __host__ __forceinline__ chunk& operator=(const chunk& other) {
        #ifdef __CUDA_ARCH__
            reinterpret_cast<uint4*>(data)[0] = reinterpret_cast<const uint4*>(other.data)[0];
            reinterpret_cast<uint4*>(data)[1] = reinterpret_cast<const uint4*>(other.data)[1];
        #else
        for (int i = 0; i < 8; i++) {
            data[i] = other.data[i];
        }
        #endif
        return *this;
    }
};

int main() {
    chunk *data, *data_d, *data_h;
    int m = 16, n = (1 << 26) / m;
    hipHostAlloc(&data, m * n * sizeof(chunk), hipHostMallocDefault);
    hipHostAlloc(&data_h, m * n * sizeof(chunk), hipHostMallocDefault);
    for (int i = 0; i < m; i++) {
        for (int j = 0; j < n; j++) {
            for (int k = 0; k < 8; k++) {
                data[IDX(i, j, n)].data[k] = i * n + j + k;
            }
        }
    }
    memcpy(data_h, data, sizeof(chunk) * m * n);

    hipMalloc(&data_d, sizeof(chunk) * m * n);
    hipMemcpy(data_d, data, sizeof(chunk) * m * n, hipMemcpyHostToDevice);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    inplace::transpose(true, data_d, m, n);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Time: %f ms\n", milliseconds);

    hipMemcpy(data, data_d, sizeof(chunk) * m * n, hipMemcpyDeviceToHost);

    for (int i = 0; i < n; i++) {
        for (int j = 0; j < m; j++) {
            for (int k = 0; k < 8; k++) {
                assert(data[IDX(i, j, m)].data[k] == data_h[IDX(j, i, n)].data[k]);
            }
        }
    }

    hipFree(data_d);
    hipHostFree(data);
    return 0;
}