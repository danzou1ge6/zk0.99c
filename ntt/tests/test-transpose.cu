#include "../src/inplace_transpose/cuda/transpose.cuh"
#include <cstring>
#include <hip/hip_runtime.h>
#define IDX(i, j, n) ((i) * (n) + (j))
struct __align__(16) chunk {
    int data[8];
};

int main() {
    chunk *data, *data_d, *data_h;
    int m = 16, n = (1 << 26) / m;
    hipHostAlloc(&data, m * n * sizeof(chunk), hipHostMallocDefault);
    hipHostAlloc(&data_h, m * n * sizeof(chunk), hipHostMallocDefault);
    for (int i = 0; i < m; i++) {
        for (int j = 0; j < n; j++) {
            for (int k = 0; k < 8; k++) {
                data[IDX(i, j, n)].data[k] = i * n + j + k;
            }
        }
    }
    memcpy(data_h, data, sizeof(chunk) * m * n);

    hipMalloc(&data_d, sizeof(chunk) * m * n);
    hipMemcpy(data_d, data, sizeof(chunk) * m * n, hipMemcpyHostToDevice);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    inplace::transpose(true, data_d, m, n);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Time: %f ms\n", milliseconds);

    hipMemcpy(data, data_d, sizeof(chunk) * m * n, hipMemcpyDeviceToHost);

    for (int i = 0; i < n; i++) {
        for (int j = 0; j < m; j++) {
            for (int k = 0; k < 8; k++) {
                assert(data[IDX(i, j, m)].data[k] == data_h[IDX(j, i, n)].data[k]);
            }
        }
    }

    hipFree(data_d);
    hipHostFree(data);
    return 0;
}