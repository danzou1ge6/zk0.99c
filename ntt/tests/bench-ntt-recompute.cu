#include "../src/recompute_ntt.cuh"
#include "../../mont/src/bn254_fr.cuh"
#include "../src/cooley_turkey_ntt.cuh"

using namespace ntt;
typedef bn254_fr::Element Field;

int main () {
    for (int k = 20; k <= 28; k += 2) {
        auto omega = Field::host_random();

        cooley_turkey_ntt<Field> ntt(reinterpret_cast<u32*>(&omega), k, false);
        ntt.to_gpu();
        
        Field *data, *data_d;
        hipMalloc(&data_d, (1ll << k) * sizeof(Field));
        data = (Field*)malloc((1ll << k) * sizeof(Field));
        for (int i = 0; i < (1ll << k); i++) {
            data[i] = Field::host_random();
        }
        hipMemcpy(data_d, data, (1ll << k) * sizeof(Field), hipMemcpyHostToDevice);

        // warm up, because the jit compilation is slow
        // for (int i = 0; i < 10; i++) ntt.ntt(reinterpret_cast<u32*>(data_d), 0, 0, true);

        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);

        for (int i = 0; i < 20; i++) {
            hipEventRecord(start);
            ntt.ntt(reinterpret_cast<u32*>(data_d), 0, 0, true);
            hipEventRecord(stop);
            hipEventSynchronize(stop);
            float milliseconds = 0;
            hipEventElapsedTime(&milliseconds, start, stop);
            printf("k = %d, time = %f ms\n", k, milliseconds);
        }
        
        free(data);
        hipFree(data_d);
    }
    return 0;
}
