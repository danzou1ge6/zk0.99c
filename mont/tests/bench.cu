#include "hip/hip_runtime.h"
#include "../src/bn256_fr.cuh"
#include "../src/field_tc.cuh"

#include <iostream>

using bn256_fr::Element;
using mont::u32;

const u32 BATCH = 128;
const u32 THREADS = 512;
const u32 ITERS = 2;

__global__ void bench(Element *r, const Element *a, const Element *b)
{
  Element v[4];
  for (u32 j = 0; j < 4; j++)
    v[j] = b[j];
  for (u32 i = 0; i < BATCH; i++)
    for (u32 j = 0; j < 4; j++)
      v[j] = v[j] * *a;
  for (u32 j = 0; j < 4; j++)
    r[j] = v[j];
}

__global__ void bench_bn(Element *r, const Element *a, const Element *b)
{
  Element v[4];
  for (u32 j = 0; j < 4; j++)
    v[j] = b[j];
  for (u32 i = 0; i < BATCH; i++)
    for (u32 j = 0; j < 4; j++)
    {
      auto prod = v[j].n * a->n;
      memcpy(v[j].n.limbs, prod.limbs, 8 * sizeof(u32));
    }
  for (u32 j = 0; j < 4; j++)
    r[j] = v[j];
}

using mont::tc256::debug::Intermediates;

template <bool DEBUG>
__global__ void bench_tc(Element *r, const Element *a, const Element *b, Intermediates *di)
{
  using namespace mont::tc256;
  u32 lane_id = threadIdx.x % 32;
  u32 warp_id = threadIdx.x / 32;

  __shared__ ConstantLoader<bn256_fr::Params> cl;
  if (warp_id == 0)
    cl.load();
  
  Multiplier mul(cl);

  __shared__ FragmentA fa[THREADS / 32];
  fa[warp_id].load(a->n.limbs);

  auto fb = FragmentB::load<0b1111>([b](u32 i, u32 j)
                                    { return b[i].n.limbs[j]; });

  FragmentW fr;
  if (DEBUG)
  {
    for (u32 i = 0; i < BATCH; i++)
    {
      if (warp_id == 0 && blockIdx.x == 0)
        fr = mul.template execute<true>(fa[warp_id], fb, di);
      else
        fr = mul(fa[warp_id], fb);
      fb = fr.transpose_to_b();
    }
  }
  else
  {
    for (u32 i = 0; i < BATCH; i++)
    {
      fr = mul(fa[warp_id], fb);
      fb = fr.transpose_to_b();
    }
  }

  if (warp_id == 0 && blockIdx.x == 0)
    fr.store<0b1111>([r](u32 i, u32 j, u32 w)
                     { r[i].n.limbs[j] = w; });
}

template <bool DEBUG>
__global__ void bench_bn_tc(Element *r, const Element *a, const Element *b, Intermediates *di)
{
  using namespace mont::tc256;
  u32 lane_id = threadIdx.x % 32;
  u32 warp_id = threadIdx.x / 32;

  __shared__ FragmentA fa[THREADS / 32];
  fa[warp_id].load(a->n.limbs);

  auto fb = FragmentB::load<0b1111>([b](u32 i, u32 j)
                                    { return b[i].n.limbs[j]; });

  FragmentW fr;
  if (DEBUG)
  {
    for (u32 i = 0; i < BATCH; i++)
    {
      if (warp_id == 0 && blockIdx.x == 0)
        fr = number_multiplication<true>(fa[warp_id], fb, di);
      else
        fr = number_multiplication<false>(fa[warp_id], fb);
      fb = fr.transpose_to_b();
    }
  }
  else
  {
    for (u32 i = 0; i < BATCH; i++)
    {
      fr = number_multiplication<false>(fa[warp_id], fb);
      fb = fr.transpose_to_b();
    }
  }

  if (warp_id == 0 && blockIdx.x == 0)
    fr.store<0b1111>([r](u32 i, u32 j, u32 w)
                     { r[i].n.limbs[j] = w; });
}

template <typename F, typename F1>
float time_it(u32 iters, F f, F1 op, bool print_intermediates, bool check)
{
  float total_time = 0;

  Element *r, *a, *b;
  hipMalloc(&r, sizeof(Element) * 4);
  hipMalloc(&a, sizeof(Element));
  hipMalloc(&b, sizeof(Element) * 4);

  auto intermediates = Intermediates::new_device();
  Intermediates *d_intermediates;
  hipMalloc(&d_intermediates, sizeof(Intermediates));
  hipMemcpy(d_intermediates, &intermediates, sizeof(Intermediates), hipMemcpyHostToDevice);

  for (u32 i = 0; i < iters; i++)
  {
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    auto ha = Element::host_random();
    Element hb[4];
    for (u32 i = 0; i < 4; i++)
      hb[i] = Element::host_random();
    hipMemcpy(a, &ha, sizeof(Element), hipMemcpyHostToDevice);
    hipMemcpy(b, hb, sizeof(Element) * 4, hipMemcpyHostToDevice);

    hipEventRecord(start);
    f(r, a, b, d_intermediates);
    hipEventRecord(stop);

    auto err = hipDeviceSynchronize();
    if (err != hipSuccess)
    {
      std::cout << "CUDA error: " << hipGetErrorString(err) << std::endl;
      std::exit(1);
    }

    Element hv[4] = {hb[0], hb[1], hb[2], hb[3]};
    for (u32 i = 0; i < BATCH; i++)
      for (u32 j = 0; j < 4; j++)
        hv[j] = op(hv[j], ha);

    Element hr[4];
    hipMemcpy(hr, r, sizeof(Element) * 4, hipMemcpyDeviceToHost);

    for (u32 j = 0; j < 4; j++)
      if (hr[j] != hv[j] & check)
      {
        std::cout << "Computation error at iteration " << std::dec << i << " : "
                  << ha.n << " ^ " << std::dec << BATCH << " * " << hb[j].n << " = " << hv[j].n
                  << ", but got " << hr[j] << std::endl;
        if (print_intermediates)
          std::cout << "Intermediates:" << std::endl
                    << intermediates.to_host();
        // std::exit(1);
      }

    hipEventSynchronize(stop);
    float elapsed_time;
    hipEventElapsedTime(&elapsed_time, start, stop);

    total_time += elapsed_time;
  }

  hipFree(r);
  hipFree(a);
  hipFree(b);

  return total_time;
}

int main(int argc, char *argv[])
{
  auto in_args = [argc, argv](const char* s)
  {
    for (int i = 1; i < argc; i ++)
      if (strcmp(s, argv[i]) == 0)
        return true;
    return false;
  };

  bool debug = in_args("debug");
  bool correctness_check = in_args("check");
  
  if (debug)
    std::cout << "Debug mode is on" << std::endl;

  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, 0);

  u32 grid_size = 8 * deviceProp.multiProcessorCount;

  std::cout << "Montgomery Multiplication" << std::endl;

  auto mmul = [](Element &x, Element &y)
  { return x * y; };

  float total_time = time_it(ITERS, [grid_size](Element *r, Element *a, Element *b, Intermediates *di)
                             { bench<<<grid_size, THREADS>>>(r, a, b); }, mmul, false, correctness_check);
  std::cout << "CUDA Core  : " << THREADS * 4 * ITERS * BATCH * grid_size / total_time * 1000 << std::endl;

  float total_time_tc = time_it(ITERS, [grid_size, debug](Element *r, Element *a, Element *b, Intermediates *di)
                                { if (debug) bench_tc<true><<<grid_size, THREADS>>>(r, a, b, di);
                                  else bench_tc<false><<<grid_size, THREADS>>>(r, a, b, di); }, mmul, debug, correctness_check);
  std::cout << "Tensor Core: " << THREADS / 8 * ITERS * BATCH * grid_size / total_time_tc * 1000 << std::endl;

  std::cout << "Big Number Multiplication" << std::endl;

  auto bnmul = [](Element &x, Element &y)
  {
    auto prod = x.n * y.n;
    Element r;
    memcpy(r.n.limbs, prod.limbs, 8 * sizeof(u32));
    return r;
  };

  float total_time_bn = time_it(ITERS, [grid_size](Element *r, Element *a, Element *b, Intermediates *di)
                                { bench_bn<<<grid_size, THREADS>>>(r, a, b); }, bnmul, false, correctness_check);
  std::cout << "CUDA Core  : " << THREADS * 4 * ITERS * BATCH * grid_size / total_time_bn * 1000 << std::endl;

  float total_time_bn_tc = time_it(ITERS, [grid_size, debug](Element *r, Element *a, Element *b, Intermediates *di)
                                   { if (debug) bench_bn_tc<true><<<grid_size, THREADS>>>(r, a, b, di);
                                           else bench_bn_tc<false><<<grid_size, THREADS>>>(r, a, b, di); }, bnmul, debug, correctness_check);
  std::cout << "Tensor Core: " << THREADS / 8 * ITERS * BATCH * grid_size / total_time_bn_tc * 1000 << std::endl;
}
