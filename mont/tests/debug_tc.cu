#include "hip/hip_runtime.h"
#include "../src/bn256_fr.cuh"
#include "../src/field_tc.cuh"

#include <iostream>

using bn256_fr::Element;

__global__ void kernel(const Element *x, const Element *y, Element *z, mont::tc256::debug::Intermediates *i)
{
  const mont::Reference st_y[4] = {
      mont::Reference((mont::u32 *)y),
      mont::Reference((mont::u32 *)(y + 1)),
      mont::Reference((mont::u32 *)(y + 2)),
      mont::Reference((mont::u32 *)(y + 3)),
  };
  mont::Reference st_z[4] = {
      mont::Reference((mont::u32 *)z),
      mont::Reference((mont::u32 *)(z + 1)),
      mont::Reference((mont::u32 *)(z + 2)),
      mont::Reference((mont::u32 *)(z + 3)),
  };
  using namespace mont::tc256;

  FragmentA fx(x->n.limbs.to_ref());
  auto fy = FragmentB::load<0b1111>(st_y);
  auto fz = mul<bn256_fr::Params, true>(fx, fy, i);
  fz.store<0b1111>(st_z);
}

int main()
{
  Element x = mont::Number<8>(BIG_INTEGER_CHUNKS8(0x06074b4b, 0x1df79173, 0x3c133ef9, 0x1819d4bc, 0xd33fac94, 0xe36715f1, 0x7779c165, 0xd12e658d));
  Element y[4] = {
      mont::Number<8>(BIG_INTEGER_CHUNKS8(0x1457b41b, 0xc2455063, 0x1b0a7958, 0xa4803a05, 0x755211e3, 0xa13bbbd6, 0x5be452ae, 0x7e785885)),
      mont::Number<8>(BIG_INTEGER_CHUNKS8(0x1457b41b, 0xc2455063, 0x1b0a7958, 0xa4803a05, 0x755211e3, 0xa13bbbd6, 0x5be452ae, 0x7e785885)),
      mont::Number<8>(BIG_INTEGER_CHUNKS8(0x1457b41b, 0xc2455063, 0x1b0a7958, 0xa4803a05, 0x755211e3, 0xa13bbbd6, 0x5be452ae, 0x7e785885)),
      mont::Number<8>(BIG_INTEGER_CHUNKS8(0x1457b41b, 0xc2455063, 0x1b0a7958, 0xa4803a05, 0x755211e3, 0xa13bbbd6, 0x5be452ae, 0x7e785885)),
  };

  // Element x = mont::Number<8>(BIG_INTEGER_CHUNKS8(0x01010101, 0x01010101, 0x01010101, 0x01010101, 0x01010101, 0x01010101, 0x01010101, 0x01010101));
  // Element y[4] = {
  //     mont::Number<8>(BIG_INTEGER_CHUNKS8(0x01010101, 0x01010101, 0x01010101, 0x01010101, 0x01010101, 0x01010101, 0x01010101, 0x01010101)),
  //     mont::Number<8>(BIG_INTEGER_CHUNKS8(0x01010101, 0x01010101, 0x01010101, 0x01010101, 0x01010101, 0x01010101, 0x01010101, 0x01010101)),
  //     mont::Number<8>(BIG_INTEGER_CHUNKS8(0x01010101, 0x01010101, 0x01010101, 0x01010101, 0x01010101, 0x01010101, 0x01010101, 0x01010101)),
  //     mont::Number<8>(BIG_INTEGER_CHUNKS8(0x01010101, 0x01010101, 0x01010101, 0x01010101, 0x01010101, 0x01010101, 0x01010101, 0x01010101)),
  // };

  Element z[4];
  auto i = mont::tc256::debug::Intermediates::new_device();

  Element *dx, *dy, *dz;
  mont::tc256::debug::Intermediates *di;
  hipMalloc(&dx, sizeof(Element));
  hipMalloc(&dy, sizeof(Element) * 4);
  hipMalloc(&dz, sizeof(Element) * 4);
  hipMalloc(&di, sizeof(mont::tc256::debug::Intermediates));
  hipMemcpy(dx, &x, sizeof(Element), hipMemcpyHostToDevice);
  hipMemcpy(dy, y, sizeof(Element) * 4, hipMemcpyHostToDevice);
  hipMemcpy(di, &i, sizeof(mont::tc256::debug::Intermediates), hipMemcpyHostToDevice);

  kernel<<<1, 32>>>(dx, dy, dz, di);
  auto err = hipStreamSynchronize(0);
  if (err != hipSuccess)
  {
    std::cout << "CUDA error: " << hipGetErrorString(err) << std::endl;
    return 1;
  }

  hipMemcpy(&z, dz, sizeof(Element) * 4, hipMemcpyDeviceToHost);

  std::cout << "Correct answer = " << (x * y[0]).n << std::endl;
  std::cout << "Got            = " << z[0].n << std::endl;
  std::cout << i.to_host();

  return 0;
}
