#include "hip/hip_runtime.h"
#include "../src/bn256_fr.cuh"
#include "../src/field_tc.cuh"

#include <iostream>

using bn256_fr::Element;

__global__ void kernel(const Element *x, const Element *y, Element *z, mont::tc256::debug::Intermediates *i)
{
  // const mont::Reference st_y[4] = {
  //     mont::Reference((mont::u32 *)y),
  //     mont::Reference((mont::u32 *)(y + 1)),
  //     mont::Reference((mont::u32 *)(y + 2)),
  //     mont::Reference((mont::u32 *)(y + 3)),
  // };
  // mont::Reference st_z[4] = {
  //     mont::Reference((mont::u32 *)z),
  //     mont::Reference((mont::u32 *)(z + 1)),
  //     mont::Reference((mont::u32 *)(z + 2)),
  //     mont::Reference((mont::u32 *)(z + 3)),
  // };
  // const mont::Reference st_x = mont::Reference((mont::u32 *)x);
  // mont::tc256::mul<4, true, bn256_fr::Params>(st_z, st_x, st_y, i);
  using namespace mont::tc256;
  using mont::u32;

  u32 a0, a1, a2, a3;
  debug::polulate_a_matrix(a0, a1, a2, a3, [](u32 i, u32 j) { return j; });
  u32 b0, b1;
  debug::polulate_b_matrix(b0, b1, [](u32 i, u32 j) { return (i == j) ? 1 : 0; });
  u32 d0, d1, d2, d3;
  mma_m16n8k32(d0, d1, d2, d3, a0, a1, a2, a3, b0, b1, 0, 0, 0, 0);
  debug::store_a_matrix(a0, a1, a2, a3, i->xa0);
  debug::store_b_matrix(b0, b1, i->yb);
  debug::store_d_matrix(d0, d1, d2, d3, i->sd0);
}

int main()
{
  Element x = mont::Number<8>(BIG_INTEGER_CHUNKS8(1, 1, 1, 1, 1, 1, 1, 1));
  Element y[4] = {
      mont::Number<8>(BIG_INTEGER_CHUNKS8(1, 1, 1, 1, 1, 1, 1, 1)),
      mont::Number<8>(BIG_INTEGER_CHUNKS8(1, 1, 1, 1, 1, 1, 1, 1)),
      mont::Number<8>(BIG_INTEGER_CHUNKS8(1, 1, 1, 1, 1, 1, 1, 1)),
      mont::Number<8>(BIG_INTEGER_CHUNKS8(1, 1, 1, 1, 1, 1, 1, 1)),
  };
  Element z[4];
  auto i = mont::tc256::debug::Intermediates::new_device();

  Element *dx, *dy, *dz;
  mont::tc256::debug::Intermediates *di;
  hipMalloc(&dx, sizeof(Element));
  hipMalloc(&dy, sizeof(Element) * 4);
  hipMalloc(&dz, sizeof(Element) * 4);
  hipMalloc(&di, sizeof(mont::tc256::debug::Intermediates));
  hipMemcpy(dx, &x, sizeof(Element), hipMemcpyHostToDevice);
  hipMemcpy(dy, y, sizeof(Element) * 4, hipMemcpyHostToDevice);
  hipMemcpy(di, &i, sizeof(mont::tc256::debug::Intermediates), hipMemcpyHostToDevice);

  kernel<<<1, 32>>>(dx, dy, dz, di);
  auto err = hipStreamSynchronize(0);
  if (err != hipSuccess)
  {
    std::cout << "CUDA error: " << hipGetErrorString(err) << std::endl;
    return 1;
  }

  hipMemcpy(&z, dz, sizeof(Element) * 4, hipMemcpyDeviceToHost);

  std::cout << z[0] << std::endl;
  std::cout << i.to_host();

  return 0;
}
