#include <iostream>
#include <fstream>

#include "../src/msm.cuh"

#if defined(CURVE_BN254)
#include "../src/bn254.cuh"
using bn254::Point;
using bn254::PointAffine;
using bn254::PointAll;
using bn254::PointAffineAll;
using Number = mont::Number<8>;

#elif defined(CURVE_BLS12381)
#include "../src/bls12381.cuh"
using bls12381::Point;
using bls12381::PointAffine;
using bls12381::PointAll;
using bls12381::PointAffineAll;
using Number = mont::Number<8>;

#elif defined(CURVE_MNT4753)
#include "../src/mnt4753.cuh"
using mnt4753::Point;
using mnt4753::PointAffine;
using mnt4753::PointAll;
using mnt4753::PointAffineAll;
using Number = mont::Number<24>;
#endif

using mont::u32;
using mont::u64;

#ifndef WINDOW_S
#define WINDOW_S 16
#endif

#ifndef ALPHA
#define ALPHA 16
#endif

#ifndef BATCH_SIZE
#define BATCH_SIZE 1
#endif

#ifndef BATCH_PER_RUN
#define BATCH_PER_RUN 1
#endif

#ifndef PARTS
#define PARTS 2
#endif

struct MsmProblem
{
  u64 len;
  PointAffineAll *points;
  Number *scalers;
};

std::istream &
operator>>(std::istream &is, MsmProblem &msm)
{
  is >> msm.len;
  msm.scalers = new Number[msm.len];
  msm.points = new PointAffineAll[msm.len];
  for (u32 i = 0; i < msm.len; i++)
  {
    char _;
    is >> msm.scalers[i] >> _ >> msm.points[i];
  }
  return is;
}

// std::ostream &
// operator<<(std::ostream &os, const MsmProblem &msm)
// {

//   for (u32 i = 0; i < msm.len; i++)
//   {
//     os << msm.scalers[i].n << '|' << msm.points[i] << std::endl;
//   }
//   return os;
// }

int main(int argc, char *argv[])
{
  if (argc != 2)
  {
    std::cout << "usage: <prog> input_file" << std::endl;
    return 2;
  }

  std::ifstream rf(argv[1]);
  if (!rf.is_open())
  {
    std::cout << "open file " << argv[1] << " failed" << std::endl;
    return 3;
  }

  MsmProblem msm;

  rf >> msm;

  hipHostRegister((void*)msm.scalers, msm.len * sizeof(Number), hipHostRegisterDefault);
  hipHostRegister((void*)msm.points, msm.len * sizeof(PointAffineAll), hipHostRegisterDefault);
#if defined(CURVE_BN254)
  using Config = msm::MsmConfig<255, WINDOW_S, ALPHA, false, TPI>;
#elif defined(CURVE_BLS12381)
  using Config = msm::MsmConfig<255, WINDOW_S, ALPHA, false, TPI>;
#elif defined(CURVE_MNT4753)
  using Config = msm::MsmConfig<753, WINDOW_S, ALPHA, false, TPI>;
#endif
  u32 stage_scalers = 2;
  u32 stage_points = 2;

  std::array<u32*, Config::n_precompute> h_points;
  h_points[0] = (u32*)msm.points;
  for (u32 i = 1; i < Config::n_precompute; i++) {
    hipHostAlloc(&h_points[i], msm.len * sizeof(PointAffineAll), hipHostMallocDefault);
  }

  
  std::vector<u32*> scalers_batches;
  for (int i = 0; i < BATCH_SIZE; i++) {
    scalers_batches.push_back((u32*)msm.scalers);
  }

  std::vector<PointAll> r(BATCH_SIZE);

  std::vector<u32> cards;
  int card_count;
  hipGetDeviceCount(&card_count);
  for (int i = 0; i < card_count; i++) {
    cards.push_back(i);
  }

  msm::MultiGPUMSM<Config, Number, Point, PointAffine, PointAll, PointAffineAll> msm_solver(msm.len, BATCH_PER_RUN, PARTS, stage_scalers, stage_points, cards);

  std::cout << "start precompute" << std::endl;

  hipStream_t stream;
  hipStreamCreate(&stream);
  msm::MSMPrecompute<Config, Point, PointAffine, PointAffineAll>::precompute(msm.len, h_points, 4);
  msm_solver.set_points(h_points);

  std::cout << "Precompute done" << std::endl;
  msm_solver.alloc_gpu();
  std::cout << "Alloc GPU done" << std::endl;
  hipEvent_t start, stop;
  float elapsedTime = 0.0;

  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);

  msm_solver.msm(scalers_batches, r);

  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsedTime, start, stop);
  std::cout << "Run done" << std::endl;

  hipStreamDestroy(stream);

  for (int i = 0; i < BATCH_SIZE; i++) {
    std::cout << r[i].to_affine() << std::endl;
  }

  std::cout << "window_size:0x" << Config::s << " alpha:0x" << Config::n_windows << " parts:0x" << PARTS << " batchs_per_run:0x" << BATCH_PER_RUN << std::endl;
  std::cout << "Total cost time:" << elapsedTime << std::endl;
  hipEventDestroy(start);
  hipEventDestroy(stop);

  hipHostUnregister((void*)msm.scalers);
  hipHostUnregister((void*)msm.points);
  for (u32 i = 1; i < Config::n_precompute; i++) {
    hipHostFree(h_points[i]);
  }

  return 0;
}