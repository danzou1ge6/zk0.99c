#include "../src/bn254.cuh"
#include "../src/msm.cuh"
#include "../../mont/src/bn254_scalar.cuh"

#include <iostream>
#include <fstream>

using bn254::Point;
using bn254::PointAffine;
using bn254_scalar::Element;
using bn254_scalar::Number;
using mont::u32;
using mont::u64;

struct MsmProblem
{
  u32 len;
  PointAffine *points;
  Element *scalers;
};

std::istream &
operator>>(std::istream &is, MsmProblem &msm)
{
  is >> msm.len;
  msm.scalers = new Element[msm.len];
  msm.points = new PointAffine[msm.len];
  for (u32 i = 0; i < msm.len; i++)
  {
    char _;
    is >> msm.scalers[i].n >> _ >> msm.points[i];
  }
  return is;
}

std::ostream &
operator<<(std::ostream &os, const MsmProblem &msm)
{

  for (u32 i = 0; i < msm.len; i++)
  {
    os << msm.scalers[i].n << '|' << msm.points[i] << std::endl;
  }
  return os;
}

int main(int argc, char *argv[])
{
  if (argc != 2)
  {
    std::cout << "usage: <prog> input_file" << std::endl;
    return 2;
  }

  std::ifstream rf(argv[1]);
  if (!rf.is_open())
  {
    std::cout << "open file " << argv[1] << " failed" << std::endl;
    return 3;
  }

  MsmProblem msm;

  rf >> msm;

  hipHostRegister((void*)msm.scalers, msm.len * sizeof(Element), hipHostRegisterDefault);
  hipHostRegister((void*)msm.points, msm.len * sizeof(PointAffine), hipHostRegisterDefault);

  using Config = msm::MsmConfig<255, 22, 2, false>;
  u32 batch_size = 4;
  u32 batch_per_run = 2;
  u32 parts = 8;
  u32 stage_scalers = 2;
  u32 stage_points = 2;

  std::array<u32*, Config::n_precompute> h_points;
  h_points[0] = (u32*)msm.points;
  for (u32 i = 1; i < Config::n_precompute; i++) {
    hipHostAlloc(&h_points[i], msm.len * sizeof(PointAffine), hipHostMallocDefault);
  }

  
  std::vector<u32*> scalers_batches;
  for (int i = 0; i < batch_size; i++) {
    scalers_batches.push_back((u32*)msm.scalers);
  }

  std::vector<Point> r(batch_size);

  std::vector<u32> cards;
  int card_count;
  hipGetDeviceCount(&card_count);
  for (int i = 0; i < card_count; i++) {
    cards.push_back(i);
  }

  msm::MultiGPUMSM<Config, Number, Point, PointAffine> msm_solver(msm.len, batch_per_run, parts, stage_scalers, stage_points, cards);

  std::cout << "start precompute" << std::endl;

  hipStream_t stream;
  hipStreamCreate(&stream);
  msm::MSMPrecompute<Config, Point, PointAffine>::precompute(msm.len, h_points, 4);
  msm_solver.set_points(h_points);

  std::cout << "Precompute done" << std::endl;
  msm_solver.alloc_gpu();
  std::cout << "Alloc GPU done" << std::endl;
  hipEvent_t start, stop;
  float elapsedTime = 0.0;

  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);

  msm_solver.msm(scalers_batches, r);

  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsedTime, start, stop);
  std::cout << "Run done" << std::endl;

  hipStreamDestroy(stream);

  for (int i = 0; i < batch_size; i++) {
    std::cout << r[i].to_affine() << std::endl;
  }

  std::cout << "Total cost time:" << elapsedTime << std::endl;
  hipEventDestroy(start);
  hipEventDestroy(stop);

  hipHostUnregister((void*)msm.scalers);
  hipHostUnregister((void*)msm.points);
  for (u32 i = 1; i < Config::n_precompute; i++) {
    hipHostFree(h_points[i]);
  }

  return 0;
}