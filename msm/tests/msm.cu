#include "../src/bn254.cuh"
#include "../src/msm_radix_sort.cuh"
#include "../../mont/src/bn254_scalar.cuh"

#include <iostream>
#include <fstream>

using bn254::Point;
using bn254::PointAffine;
using bn254_scalar::Element;
using bn254_scalar::Number;
using mont::u32;
using mont::u64;

struct MsmProblem
{
  u32 len;
  PointAffine *points;
  Element *scalers;
};

std::istream &
operator>>(std::istream &is, MsmProblem &msm)
{
  is >> msm.len;
  msm.scalers = new Element[msm.len];
  msm.points = new PointAffine[msm.len];
  for (u32 i = 0; i < msm.len; i++)
  {
    char _;
    is >> msm.scalers[i].n >> _ >> msm.points[i];
  }
  return is;
}

std::ostream &
operator<<(std::ostream &os, const MsmProblem &msm)
{

  for (u32 i = 0; i < msm.len; i++)
  {
    os << msm.scalers[i].n << '|' << msm.points[i] << std::endl;
  }
  return os;
}

int main(int argc, char *argv[])
{
  if (argc != 2)
  {
    std::cout << "usage: <prog> input_file" << std::endl;
    return 2;
  }

  std::ifstream rf(argv[1]);
  if (!rf.is_open())
  {
    std::cout << "open file " << argv[1] << " failed" << std::endl;
    return 3;
  }

  MsmProblem msm;

  rf >> msm;

  hipHostRegister((void*)msm.scalers, msm.len * sizeof(Element), hipHostRegisterDefault);
  hipHostRegister((void*)msm.points, msm.len * sizeof(PointAffine), hipHostRegisterDefault);

  u32 *d_points, head;

  using Config = msm::MsmConfig<>;

  u32 *h_points[Config::n_precompute];
  h_points[0] = (u32*)msm.points;
  for (u32 i = 1; i < Config::n_precompute; i++) {
    hipHostAlloc(&h_points[i], msm.len * sizeof(PointAffine), hipHostMallocDefault);
  }

  hipStream_t stream;
  hipStreamCreate(&stream);

  msm::precompute<Config>(h_points, msm.len, stream);

  hipEvent_t start, stop;
  float elapsedTime = 0.0;

  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);

  Point r;
  msm::run<Config>(msm.len, (u32*)msm.scalers, const_cast<const u32 **>(h_points), r, false, false, d_points, head, stream);

  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsedTime, start, stop);

  hipStreamDestroy(stream);

  std::cout << r.to_affine() << std::endl;

  std::cout << "Total cost time:" << elapsedTime << std::endl;
  hipEventDestroy(start);
  hipEventDestroy(stop);

  hipHostUnregister((void*)msm.scalers);
  hipHostUnregister((void*)msm.points);
  for (u32 i = 1; i < Config::n_precompute; i++) {
    hipHostFree(h_points[i]);
  }

  return 0;
}